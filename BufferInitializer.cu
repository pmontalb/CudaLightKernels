#include "BufferInitializer.cuh"
#include "DeviceManager.cuh"
#include <stdio.h>

EXTERN_C
{
	EXPORT int _Initialize(MemoryBuffer buf, const double value)
	{
		switch (buf.mathDomain)
		{
		case MathDomain::Float:
			CUDA_CALL_SINGLE(__Initialize__<float>, (float*)buf.pointer, buf.size, (float)value);
			break;
		case MathDomain::Double:
			CUDA_CALL_DOUBLE(__Initialize__<double>, (double*)buf.pointer, buf.size, value);
			break;
		case MathDomain::Int:
			CUDA_CALL_SINGLE(__Initialize__<int>, (int*)buf.pointer, buf.size, (int)value);
			break;
		default:
			return -1;
		}

		return hipGetLastError();
	}

	EXPORT int _LinSpace(MemoryBuffer buf, const double x0, const double x1)
	{
		const double dx = (x1 - x0) / (buf.size - 1);

		switch (buf.mathDomain)
		{
		case MathDomain::Float:
			CUDA_CALL_SINGLE(__LinSpace__<float>, (float*)buf.pointer, buf.size, (float)x0, (float)dx);
			break;
		case MathDomain::Double:
			CUDA_CALL_DOUBLE(__LinSpace__<double>, (double*)buf.pointer, buf.size, (double)x0, (double)dx);
			break;
		default:
			return -1;
		}

		return hipGetLastError();
	}

	EXPORT int _RandUniform(MemoryBuffer buf, const unsigned seed)
	{
		if (buf.size & 1)
			return -1;

		dim3 block, grid;
		const unsigned halfSz = buf.size >> 1;
		detail::GetBestDimension(block, grid, N_BLOCKS_SINGLE, halfSz);

		hiprandState *states = 0;
		if (hipMalloc((void **)&states, grid.x * block.x * sizeof(hiprandState)))
			return -1;
		CUDA_CALL_XY(__SetupCuRand__, grid, block, states, halfSz, seed);

		switch (buf.mathDomain)
		{
		case MathDomain::Float:
			CUDA_CALL_XYZ(__RandUniform__<float>, grid, block, block.x * sizeof(unsigned int), (float*)buf.pointer, states, halfSz);
			break;
		case MathDomain::Double:
			CUDA_CALL_XYZ(__RandUniform__<double>, grid, block, block.x * sizeof(unsigned int), (double*)buf.pointer, states, halfSz);
			break;
		default:
			return -1;
		}

		hipFree(states);

		return hipGetLastError();
	}

	EXPORT int _RandNormal(MemoryBuffer buf, const unsigned seed)
	{
		if (buf.size & 1)
			return -1;

		dim3 block, grid;
		const unsigned halfSz = buf.size >> 1;
		detail::GetBestDimension(block, grid, N_BLOCKS_SINGLE, halfSz);

		hiprandState *states = 0;
		if (hipMalloc((void **)&states, grid.x * block.x * sizeof(hiprandState)))
			return -1;
		CUDA_CALL_XY(__SetupCuRand__, grid, block, states, halfSz, seed);

		switch (buf.mathDomain)
		{
		case MathDomain::Float:
			CUDA_CALL_XYZ(__RandNormal__<float>, grid, block, block.x * sizeof(unsigned int), (float*)buf.pointer, states, halfSz);
			break;
		case MathDomain::Double:
			CUDA_CALL_XYZ(__RandNormal__<double>, grid, block, block.x * sizeof(unsigned int), (double*)buf.pointer, states, halfSz);
			break;
		default:
			return -1;
		}

		hipFree(states);

		return hipGetLastError();
	}

	EXPORT int _Eye(MemoryTile buf)
	{
		dim3 blockDim(16, 16);
		dim3 gridDim((buf.nRows + 15) / 16, (buf.nRows + 15) / 16);

		switch (buf.mathDomain)
		{
		case MathDomain::Float:
			CUDA_CALL_XY(__Eye__<float>, gridDim, blockDim, (float*)buf.pointer, buf.nRows);
			break;
		case MathDomain::Double:
			CUDA_CALL_XY(__Eye__<double>, gridDim, blockDim, (double*)buf.pointer, buf.nRows);
			break;
		default:
			return -1;
		}
			
		return hipGetLastError();
	}

	EXPORT int _OnesUpperTriangular(MemoryTile buf)
	{
		dim3 blockDim(16, 16);
		dim3 gridDim((buf.nRows + 15) / 16, (buf.nRows + 15) / 16);

		switch (buf.mathDomain)
		{
		case MathDomain::Float:
			CUDA_CALL_XY(__OnesUpperTriangular__<float>, gridDim, blockDim, (float*)buf.pointer, buf.nRows);
			break;
		case MathDomain::Double:
			CUDA_CALL_XY(__OnesUpperTriangular__<double>, gridDim, blockDim, (double*)buf.pointer, buf.nRows);
			break;
		default:
			return -1;
		}

		return hipGetLastError();
	}
}

template <typename T>
GLOBAL void __Initialize__(T* RESTRICT ptr, const ptr_t sz, const T value)
{
	CUDA_FUNCTION_PROLOGUE
	CUDA_FOR_LOOP_PROLOGUE

		ptr[i] = value;

	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __LinSpace__(T* RESTRICT ptr, const ptr_t sz, const T x0, const T dx)
{
	CUDA_FUNCTION_PROLOGUE
	CUDA_FOR_LOOP_PROLOGUE

		ptr[i] = x0 + i * dx;

	CUDA_FOR_LOOP_EPILOGUE
}

GLOBAL void __SetupCuRand__(CURAND_STATE_PTR states, const ptr_t sz, const unsigned seed)
{
	// Determine thread ID
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Initialise the RNG
	hiprand_init(seed, tid, 0, &states[tid]);
}

template <typename T>
GLOBAL void __RandUniform__(T* RESTRICT ptr, CURAND_STATE_PTR states, const ptr_t sz)
{
	CUDA_FUNCTION_PROLOGUE

	hiprandState localState = states[tid];

	CUDA_FOR_LOOP_PROLOGUE

		ptr[2 * i] = hiprand_uniform(&localState);
		ptr[2 * i + 1] = 1.0f - ptr[2 * i];

	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __RandNormal__(T* RESTRICT ptr, CURAND_STATE_PTR states, const ptr_t sz)
{
	CUDA_FUNCTION_PROLOGUE

	hiprandState localState = states[tid];

	CUDA_FOR_LOOP_PROLOGUE

		ptr[2 * i] = hiprand_normal(&localState);
		ptr[2 * i + 1] = -ptr[2 * i];

	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __Eye__(T* RESTRICT A, size_t sz)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	T one = static_cast<T>(1.0);
	T zero = static_cast<T>(0.0);
	if (j < sz && i < sz)
	{
		if (i == j)
			A[i + sz * j] = one;
		else
			A[i + sz * j] = zero;
	}
}

template <typename T>
GLOBAL void __OnesUpperTriangular__(T* RESTRICT A, size_t sz)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	T one = static_cast<T>(1.0);
	T zero = static_cast<T>(0.0);
	if (j < sz && i < sz)
	{
		if (i <= j)
			A[i + sz * j] = one;
		else
			A[i + sz * j] = zero;
	}
}