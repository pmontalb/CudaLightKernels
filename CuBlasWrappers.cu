#include "hip/hip_runtime.h"
#include "CuBlasWrappers.cuh"
#include "DeviceManager.cuh"
#include "BufferInitializer.cuh"
#include "MemoryManager.cuh"
#include <hipblas.h>

EXTERN_C
{
	/**
	* z = x + y
	*/
	EXPORT int _Add(MemoryBuffer& z, const MemoryBuffer& x, const MemoryBuffer& y, const double alpha)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();

		switch (z.mathDomain)
		{
		case MathDomain::Float:
			{
				int err = hipblasScopy(handle, z.size, (float*)y.pointer, 1, (float*)z.pointer, 1);
				if (err)
					return err;

				const float _alpha = (float)alpha;
				return hipblasSaxpy(handle, z.size, &_alpha, (float*)x.pointer, 1, (float*)z.pointer, 1);
			}
		case MathDomain::Double:
			{
				int err = hipblasDcopy(handle, z.size, (double*)y.pointer, 1, (double*)z.pointer, 1);
				if (err)
					return err;

				return hipblasDaxpy(handle, z.size, &alpha, (double*)x.pointer, 1, (double*)z.pointer, 1);
			}
		case MathDomain::Int:
			{
			    CUDA_CALL_SINGLE(__IntAffineOperationNaive__, (int*)z.pointer, (int*)x.pointer, (int*)y.pointer, z.size, (int)alpha, 1, 0);
				return hipGetLastError();
			}
		default:
			return CudaKernelException::_NotImplementedException;
		}
	}
	EXPORT int _AddRaw(const ptr_t z, const ptr_t x, const ptr_t y, const unsigned size, const MemorySpace memorySpace, const MathDomain mathDomain, const double alpha)
	{
		MemoryBuffer _z(z, size, memorySpace, mathDomain);
		MemoryBuffer _x(x, size, memorySpace, mathDomain);
		MemoryBuffer _y(y, size, memorySpace, mathDomain);
		return _Add(_z, _x, _y, alpha);
	 }

	/**
	* z += x
	*/
	EXPORT int _AddEqual(MemoryBuffer& z, const MemoryBuffer& x, const double alpha)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();

		switch (z.mathDomain)
		{
		case MathDomain::Float:
		{
			const float _alpha = (float)alpha;
			return hipblasSaxpy(handle, z.size, &_alpha, (float*)x.pointer, 1, (float*)z.pointer, 1);
		}
		case MathDomain::Double:
			return hipblasDaxpy(handle, z.size, &alpha, (double*)x.pointer, 1, (double*)z.pointer, 1);
		case MathDomain::Int:
			CUDA_CALL_SINGLE(__IntAffineOperationNaive__, (int*)z.pointer, (int*)x.pointer, (int*)x.pointer, z.size, 0, (int)alpha, 1);
			return hipGetLastError();
		default:
			return CudaKernelException::_NotImplementedException;
		}
	}
	EXPORT int _AddEqualRaw(const ptr_t z, const ptr_t x, const unsigned size, const MemorySpace memorySpace, const MathDomain mathDomain, const double alpha)
	{
		MemoryBuffer _z(z, size, memorySpace, mathDomain);
		MemoryBuffer _x(x, size, memorySpace, mathDomain);
		return _AddEqual(_z, _x, alpha);
    }

    EXPORT int _Subtract(MemoryBuffer& z, const MemoryBuffer& x, const MemoryBuffer& y)
    {
        return _Add(z, y, x, -1.0);
    }

    EXPORT int _SubtractEqual(MemoryBuffer& z, const MemoryBuffer& x)
    {
        return _AddEqual(z, x, -1.0);
    }

	/**
	* A += alpha * B
	*/
	EXPORT int _AddEqualMatrix(MemoryTile& A, const MemoryTile& B, const MatrixOperation aOperation, const MatrixOperation bOperation, const double alpha, const double beta)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();

		switch (A.mathDomain)
		{
		case MathDomain::Float:
		{
			const float _alpha = (float)alpha;
			const float _beta = (float)beta;
			return hipblasSgeam(handle, cublasOperation[static_cast<unsigned>(aOperation)], cublasOperation[static_cast<unsigned>(bOperation)],
				A.nRows, A.nCols,
				&_alpha,
				(float*)A.pointer, A.leadingDimension,
				&_beta,
				(float*)B.pointer, B.leadingDimension,
				(float*)A.pointer, A.leadingDimension);
		}
		case MathDomain::Double:
		{
			return hipblasDgeam(handle, cublasOperation[static_cast<unsigned>(aOperation)], cublasOperation[static_cast<unsigned>(bOperation)],
				A.nRows, A.nCols,
				&alpha,
				(double*)A.pointer, A.leadingDimension,
				&beta,
				(double*)B.pointer, B.leadingDimension,
				(double*)A.pointer, A.leadingDimension);
		}
		default:
			return CudaKernelException::_NotImplementedException;
		}
	}
	EXPORT int _AddEqualMatrixRaw(const ptr_t A, const ptr_t B, const unsigned nRows, const unsigned nCols, const MemorySpace memorySpace, const MathDomain mathDomain, const MatrixOperation aOperation, const MatrixOperation bOperation, const double alpha, const double beta)
	{
		MemoryTile _A(A, nRows, nCols, memorySpace, mathDomain);
		MemoryTile _B(B, nRows, nCols, memorySpace, mathDomain);
		return _AddEqualMatrix(_A, _B, aOperation, bOperation, alpha, beta);
	}

	/**
	* z *= alpha
	*/
	EXPORT int _Scale(MemoryBuffer& z, const double alpha)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		switch (z.mathDomain)
		{
		case MathDomain::Float:
		{
			const float _alpha = (float)alpha;
			return hipblasSscal(handle, z.size, &_alpha, (float*)z.pointer, 1);
		}
		case MathDomain::Double:
			return hipblasDscal(handle, z.size, &alpha, (double*)z.pointer, 1);
		case MathDomain::Int:
			CUDA_CALL_SINGLE(__IntAffineOperationNaive__, (int*)z.pointer, (int*)z.pointer, (int*)z.pointer, z.size, 0, 0, (int)alpha);
			return hipGetLastError();
		default:
			return CudaKernelException::_NotImplementedException;
		}
	}
	EXPORT int _ScaleRaw(const ptr_t z, const unsigned size, const MemorySpace memorySpace, const MathDomain mathDomain, const double alpha)
	{
		MemoryBuffer _z(z, size, memorySpace, mathDomain);
		return _Scale(_z, alpha);
	}

	EXPORT int _ScaleColumns(MemoryTile& z, const MemoryBuffer& alpha)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();

		hipblasPointerMode_t originalPointerMode;
		hipblasGetPointerMode(handle, &originalPointerMode);
		hipblasSetPointerMode(handle, hipblasPointerMode_t::HIPBLAS_POINTER_MODE_DEVICE);
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				for (size_t i = 0; i < z.nCols; ++i)
					hipblasSscal(handle, z.nRows, (float*)(alpha.pointer + i * alpha.ElementarySize()), (float*)(z.pointer + i * z.nRows * z.ElementarySize()), 1);
				break;
			case MathDomain::Double:
				for (size_t i = 0; i < z.nCols; ++i)
					hipblasDscal(handle, z.nRows, (double*)(alpha.pointer + i * alpha.ElementarySize()), (double*)(z.pointer + i * z.nRows * z.ElementarySize()), 1);
				break;
			default:
				hipblasSetPointerMode(handle, originalPointerMode);
				return CudaKernelException::_NotImplementedException;
		}
		hipblasSetPointerMode(handle, originalPointerMode);

		return hipGetLastError();
	}
	EXPORT int _ScaleColumnsRaw(const ptr_t z, const unsigned nRows, const unsigned nCols, const MemorySpace memorySpace, const MathDomain mathDomain, const ptr_t alpha)
	{
		MemoryTile _z(z, nRows, nCols, memorySpace, mathDomain);
		MemoryBuffer _alpha(alpha, nCols, memorySpace, mathDomain);
		return _ScaleColumns(_z, _alpha);
	}

	EXPORT int _ElementwiseProduct(MemoryBuffer& z, const MemoryBuffer& x, const MemoryBuffer& y, const double alpha)
	{
		//#define USE_NAIVE_ELEMENTWISE_PRODUCT
		#ifndef USE_NAIVE_ELEMENTWISE_PRODUCT
			const hipblasHandle_t& handle = detail::CublasHandle();

			switch (z.mathDomain)
			{
			case MathDomain::Float:
			{
				const float _alpha = (float)alpha;
				const float beta = 0.0f;

				return hipblasSsbmv(handle, HIPBLAS_FILL_MODE_UPPER,
					z.size, 0,  // Just the diagonal; 0 super-diagonal bands
					&_alpha,
					(float*)x.pointer, 1,
					(float*)y.pointer, 1,
					&beta,
					(float*)z.pointer, 1);
			}
			case MathDomain::Double:
			{
				const double beta = 0.0;
				return hipblasDsbmv(handle, HIPBLAS_FILL_MODE_UPPER,
					z.size, 0,  // Just the diagonal; 0 super-diagonal bands
					&alpha,
					(double*)x.pointer, 1,
					(double*)y.pointer, 1,
					&beta,
					(double*)z.pointer, 1);
			}
			case MathDomain::Int:
				CUDA_CALL_SINGLE(__ElementwiseProductNaive__<int COMMA false>, (int*)z.pointer, (int*)x.pointer, (int*)y.pointer, z.size, (int)alpha);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
			}
		#else

			switch (z.mathDomain)
			{
				case MathDomain::Float:
					CUDA_CALL_SINGLE(__ElementwiseProductNaive__<float COMMA false>, (float*)z.pointer, (float*)x.pointer, (float*)y.pointer, z.size, (float)alpha);
					break;
				case MathDomain::Double:
					CUDA_CALL_DOUBLE(__ElementwiseProductNaive__<double COMMA false>, (double*)z.pointer, (double*)x.pointer, (double*)y.pointer, z.size, alpha);
					break;
				case MathDomain::Int:
					CUDA_CALL_SINGLE(__ElementwiseProductNaive__<int COMMA false>, (int*)z.pointer, (int*)x.pointer, (int*)y.pointer, z.size, (int)alpha);
					break;
				default:
					return CudaKernelException::_NotImplementedException;
			}

		#endif // USE_NAIVE_ELEMENTWISE_PRODUCT

		return hipGetLastError();
	}
	EXPORT int _ElementwiseProductRaw(const ptr_t z, const ptr_t x, const ptr_t y, const unsigned size, const MemorySpace memorySpace, const MathDomain mathDomain, const double alpha)
	{
		MemoryBuffer _z(z, size, memorySpace, mathDomain);
		MemoryBuffer _x(x, size, memorySpace, mathDomain);
		MemoryBuffer _y(y, size, memorySpace, mathDomain);
		return _ElementwiseProduct(_z, _x, _y, alpha);
	}

	EXPORT int _ElementwiseDivision(MemoryBuffer& z, const MemoryBuffer& x, const MemoryBuffer& y, const double alpha)
	{
		#ifndef USE_NAIVE_ELEMENTWISE_PRODUCT
			return CudaKernelException::_NotImplementedException;
		#else

		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__ElementwiseProductNaive__<float COMMA true>, (float*)z.pointer, (float*)x.pointer, (float*)y.pointer, z.size, (float)alpha);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__ElementwiseProductNaive__<double COMMA true>, (double*)z.pointer, (double*)x.pointer, (double*)y.pointer, z.size, alpha);
				break;
			case MathDomain::Int:
				CUDA_CALL_SINGLE(__ElementwiseProductNaive__<int COMMA true>, (int*)z.pointer, (int*)x.pointer, (int*)y.pointer, z.size, (int)alpha);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}

		#endif // USE_NAIVE_ELEMENTWISE_PRODUCT

		return hipGetLastError();
	}
	EXPORT int _ElementwiseDivisionRaw(const ptr_t z, const ptr_t x, const ptr_t y, const unsigned size, const MemorySpace memorySpace, const MathDomain mathDomain, const double alpha)
	{
		MemoryBuffer _z(z, size, memorySpace, mathDomain);
		MemoryBuffer _x(x, size, memorySpace, mathDomain);
		MemoryBuffer _y(y, size, memorySpace, mathDomain);
		return _ElementwiseDivision(_z, _x, _y, alpha);
	}

	EXPORT int _SubMultiply(MemoryTile& A, const MemoryTile& B, const MemoryTile& C, const unsigned nRowsB, const unsigned nColsB, const unsigned nColsC, const MatrixOperation bOperation, const MatrixOperation cOperation, const double alpha, const double beta)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		switch (A.mathDomain)
		{
			case MathDomain::Float:
			{
				const float _alpha = (float)alpha;
				const float _beta = (float)beta;
				return hipblasSgemm(handle, cublasOperation[static_cast<unsigned>(bOperation)], cublasOperation[static_cast<unsigned>(cOperation)],
				                   nRowsB, nColsC, nColsB,
				                   &_alpha,
				                   (float*)B.pointer, B.leadingDimension,
				                   (float*)C.pointer, C.leadingDimension,
				                   &_beta,
				                   (float*)A.pointer, A.leadingDimension);
			}
			case MathDomain::Double:
			{
				return hipblasDgemm(handle, cublasOperation[static_cast<unsigned>(bOperation)], cublasOperation[static_cast<unsigned>(cOperation)],
				                   nRowsB, nColsC, nColsB,
				                   &alpha,
				                   (double*)B.pointer, B.leadingDimension,
				                   (double*)C.pointer, C.leadingDimension,
				                   &beta,
				                   (double*)A.pointer, A.leadingDimension);
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}
	}
	EXPORT int _SubMultiplyRaw(const ptr_t A, const ptr_t B, const ptr_t C, const unsigned nRowsB, const unsigned nRowsC, const unsigned nColsC, const MemorySpace memorySpace, const MathDomain mathDomain, const unsigned leadingDimensionA, const unsigned leadingDimensionB, const unsigned leadingDimensionC, const unsigned nColsB, const MatrixOperation bOperation, const MatrixOperation cOperation, const double alpha, const double beta)
	{
		MemoryTile _A(A, leadingDimensionA, nColsC, leadingDimensionA, memorySpace, mathDomain);
		MemoryTile _B(B, leadingDimensionB, nRowsC, leadingDimensionB, memorySpace, mathDomain);
		MemoryTile _C(C, leadingDimensionC, nColsC, leadingDimensionC, memorySpace, mathDomain);
		return _SubMultiply(_A, _B, _C, nRowsB, nColsB, nColsC, bOperation, cOperation, alpha, beta);
	}

	EXPORT int _Multiply(MemoryTile& A, const MemoryTile& B, const MemoryTile& C, const MatrixOperation bOperation, const MatrixOperation cOperation, const double alpha, const double beta)
	{
		return _SubMultiply(A, B, C, B.nRows, B.nCols, C.nCols, bOperation, cOperation, alpha, beta);
	}
	EXPORT int _MultiplyRaw(const ptr_t A, const ptr_t B, const ptr_t C, const unsigned nRowsB, const unsigned nRowsC, const unsigned nColsC, const MemorySpace memorySpace, const MathDomain mathDomain, const unsigned leadingDimensionB, const unsigned leadingDimensionC, const MatrixOperation bOperation, const MatrixOperation cOperation, const double alpha, const double beta)
{
	MemoryTile _A(A, nRowsB, nColsC, leadingDimensionB, memorySpace, mathDomain);
	MemoryTile _B(B, nRowsB, nRowsC, leadingDimensionB, memorySpace, mathDomain);
	MemoryTile _C(C, nRowsC, nColsC, leadingDimensionC, memorySpace, mathDomain);
	return _Multiply(_A, _B, _C, bOperation, cOperation, alpha, beta);
}

	EXPORT int _BatchedMultiply(MemoryCube& A, const MemoryCube& B, const MemoryCube& C, const unsigned strideB, const unsigned strideC, const MatrixOperation bOperation, const MatrixOperation cOperation, const double alpha, const double beta)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		switch (A.mathDomain)
		{
			case MathDomain::Float:
			{
				float _alpha = (float)alpha;
				float _beta = (float)beta;

				return hipblasSgemmStridedBatched(handle, cublasOperation[static_cast<unsigned>(bOperation)], cublasOperation[static_cast<unsigned>(cOperation)],
				                                 A.nRows, A.nCols, B.nCols,
				                                 &_alpha,
				                                 (float*)B.pointer, B.leadingDimension, strideB,
				                                 (float*)C.pointer, C.leadingDimension, strideC,
				                                 &_beta,
				                                 (float*)A.pointer, A.leadingDimension, A.nRows * A.nCols,
				                                 A.nCubes);
			}
			case MathDomain::Double:
			{
				return hipblasDgemmStridedBatched(handle, cublasOperation[static_cast<unsigned>(bOperation)], cublasOperation[static_cast<unsigned>(cOperation)],
				                                 A.nRows, A.nCols, B.nCols,
				                                 &alpha,
				                                 (double*)B.pointer, B.leadingDimension, strideB,
				                                 (double*)C.pointer, C.leadingDimension, strideC,
				                                 &beta,
				                                 (double*)A.pointer, A.leadingDimension, A.nRows * A.nCols,
				                                 A.nCubes);
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}
	}
	EXPORT int _BatchedMultiplyRaw(const ptr_t A, const ptr_t B, const ptr_t C, const unsigned nRowsB, const unsigned nRowsC, const unsigned nColsC, const unsigned nCubes, const MemorySpace memorySpace, const MathDomain mathDomain, const unsigned leadingDimensionB, const unsigned leadingDimensionC, const MatrixOperation bOperation, const MatrixOperation cOperation, const double alpha, const double beta)
	{
		MemoryCube _A(A, nRowsB, nColsC, nCubes, memorySpace, mathDomain);
		MemoryCube _B(B, nRowsB, nRowsC, nCubes, memorySpace, mathDomain);
		MemoryCube _C(C, nRowsC, nColsC, nCubes, memorySpace, mathDomain);
		return _BatchedMultiply(_A, _B, _C, _B.nRows * _B.nCols, _C.nRows * _C.nCols, bOperation, cOperation, alpha, beta);
	}

	EXPORT int _Dot(MemoryBuffer& y, const MemoryTile& A, const MemoryBuffer& x, const MatrixOperation aOperation, const double alpha, const double beta)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		switch (A.mathDomain)
		{
		case MathDomain::Float:
		{
			const float _alpha = (float)alpha;
			const float _beta = (float)beta;
			return hipblasSgemv(handle, cublasOperation[static_cast<unsigned>(aOperation)],
				A.nRows, A.nCols,
				&_alpha,
				(float*)A.pointer, A.leadingDimension,
				(float*)x.pointer, 1,
				&_beta,
				(float*)y.pointer, 1);
		}
		case MathDomain::Double:
		{
			return hipblasDgemv(handle, cublasOperation[static_cast<unsigned>(aOperation)],
				A.nRows, A.nCols,
				&alpha,
				(double*)A.pointer, A.leadingDimension,
				(double*)x.pointer, 1,
				&beta,
				(double*)y.pointer, 1);
		}
		default:
			return CudaKernelException::_NotImplementedException;
		}
	}
	EXPORT int _DotRaw(const ptr_t y, const ptr_t A, const ptr_t x, const unsigned nRows, const unsigned nCols, const MemorySpace memorySpace, const MathDomain mathDomain, const MatrixOperation aOperation, const double alpha, const double beta)
	{
		MemoryBuffer _x(x, nCols, memorySpace, mathDomain);
		MemoryBuffer _y(y, nCols, memorySpace, mathDomain);
		MemoryTile _A(A, nRows, nCols, memorySpace, mathDomain);
		return _Dot(_y, _A, _x, aOperation, alpha, beta);
	}

	EXPORT int _KroneckerProduct(MemoryTile& A, const MemoryBuffer& x, const MemoryBuffer& y, const double alpha)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		switch (A.mathDomain)
		{
			case MathDomain::Float:
			{
				const float _alpha = (float)alpha;
				return hipblasSger(handle, x.size, y.size, &_alpha, (float*)x.pointer, 1, (float*)y.pointer, 1, (float*)A.pointer, A.nRows);
			}
			case MathDomain::Double:
			{
				return hipblasDger(handle, x.size, y.size, &alpha, (double*)x.pointer, 1, (double*)y.pointer, 1, (double*)A.pointer, A.nRows);
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}
	}
	EXPORT int _KroneckerProductRaw(const ptr_t A, const ptr_t x, const ptr_t y, const unsigned nRows, const unsigned nCols, const MemorySpace memorySpace, const MathDomain mathDomain, const double alpha)
	{
		MemoryBuffer _x(x, nRows, memorySpace, mathDomain);
		MemoryBuffer _y(y, nCols, memorySpace, mathDomain);
		MemoryTile _A(A, nRows, nCols, memorySpace, mathDomain);
		return _KroneckerProduct(_A, _x, _y, alpha);
	}

	EXPORT int _BatchedTransposedKroneckerProduct(MemoryCube& T, const MemoryTile& x, const MemoryTile& y, const double alpha)
	{
		static constexpr size_t nStreams = { 32 };
		hipStream_t streams[nStreams];
		int err = 0;
		for (size_t i = 0; i < nStreams; i++)
		{
			err = hipStreamCreate(&streams[i]);
			if (err)
				return err;
		}

		const hipblasHandle_t& handle = detail::CublasHandle();
		const size_t nCubesPerStream = (T.nCubes + nStreams) / nStreams;

		size_t cubeStart = 0;
		size_t cubeEnd = nCubesPerStream;

		MemoryTile cache1(T.pointer, T.nRows, T.nCols, T.memorySpace, T.mathDomain);
		MemoryBuffer cache2(x.pointer, x.nRows, x.memorySpace, x.mathDomain);
		MemoryBuffer cache3(y.pointer, y.nRows, y.memorySpace, y.mathDomain);

		const size_t tOffset = T.nRows * T.nCols * T.ElementarySize();
		const size_t xOffset = x.nRows * x.ElementarySize();
		const size_t yOffset = y.nRows * y.ElementarySize();
		for (size_t i = 0; i < nStreams; i++)
		{
			hipblasSetStream(handle, streams[i]);
			for (size_t j = cubeStart; j < cubeEnd; ++j)
			{
				err = _KroneckerProduct(cache1, cache2, cache3, alpha);
				if (err)
					return err;

				cache1.pointer += tOffset;
				cache2.pointer += xOffset;
				cache3.pointer += yOffset;
			}

			cubeStart = cubeEnd;
			cubeEnd = min(cubeEnd + nCubesPerStream, static_cast<size_t>(T.nCubes));

			if (cubeStart == T.nCubes)
				break;
		}

		for (size_t i = 0; i < nStreams; i++)
		{
			err = hipStreamDestroy(streams[i]);
			if (err)
				return err;
		}

		// reset stream
		err = hipblasSetStream(handle, nullptr);
		if (err)
			return err;

		return hipGetLastError();
	}
	EXPORT int _BatchedTransposedKroneckerProductRaw(const ptr_t A, const ptr_t x, const ptr_t y, const unsigned nRows, const unsigned nCols, const unsigned nCubes, const MemorySpace memorySpace, const MathDomain mathDomain, const double alpha)
	{
		MemoryTile _x(x, nRows, nCubes, memorySpace, mathDomain);
		MemoryTile _y(y, nCols, nCubes, memorySpace, mathDomain);
		MemoryCube _A(A, nRows, nCols, nCubes, memorySpace, mathDomain);
		return _BatchedTransposedKroneckerProduct(_A, _x, _y, alpha);
	}

	EXPORT int _CumulativeRowSum(MemoryTile& A)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		int err = -1;

		switch(A.mathDomain)
		{
		case MathDomain::Float:
		{
			float *onesPtr = nullptr;
			err = hipMalloc((void **)&onesPtr, A.nRows * A.nCols * sizeof(float));
			if (err)
				return err;
			MemoryTile ones((ptr_t)onesPtr, A.nRows, A.nCols, A.memorySpace, A.mathDomain);
			_OnesUpperTriangular(ones);

			float *buffer = nullptr;
			err = hipMalloc((void **)&buffer, A.nRows * A.nCols * sizeof(float));
			if (err)
				return err;

			err = hipMemcpy(buffer, (void*)A.pointer, A.nRows * A.nCols * sizeof(float), hipMemcpyDeviceToDevice);
			if (err)
				return err;

			float alpha = 1.0f, beta = 0.0f;
			err = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				A.nRows, A.nCols, A.nRows,
				&alpha,
				buffer, A.leadingDimension,
				(float*)ones.pointer, A.leadingDimension,
				&beta,
				(float*)A.pointer, A.leadingDimension);

			hipFree((void*)ones.pointer);
			hipFree(buffer);
			break;
		}
		case MathDomain::Double:
		{
			double *onesPtr = nullptr;
			err = hipMalloc((void **)&onesPtr, A.nRows * A.nCols * sizeof(double));
			if (err)
				return err;

			MemoryTile ones((ptr_t)onesPtr, A.nRows, A.nCols, A.memorySpace, A.mathDomain);
			_OnesUpperTriangular(ones);

			double *buffer = nullptr;
			err = hipMalloc((void **)&buffer, A.nRows * A.nCols * sizeof(double));
			if (err)
				return err;

			hipMemcpy(buffer, (void*)A.pointer, A.nRows * A.nCols * sizeof(double), hipMemcpyDeviceToDevice);

			double alpha = 1.0, beta = 0.0;
			err = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				A.nRows, A.nCols, A.nRows,
				&alpha,
				buffer, A.leadingDimension,
				(double*)ones.pointer, A.leadingDimension,
				&beta,
				(double*)A.pointer, A.leadingDimension);

			hipFree((void*)ones.pointer);
			hipFree(buffer);
			break;
		}
		default:
			return CudaKernelException::_NotImplementedException;
		}

		return err;
	}
	EXPORT int _CumulativeRowSumRaw(const ptr_t A, const unsigned nRows, const unsigned nCols, const MemorySpace memorySpace, const MathDomain mathDomain)
	{
		MemoryTile _A(A, nRows, nCols, memorySpace, mathDomain);
		return _CumulativeRowSum(_A);
	}

	/**
	* x = sum(A[:, ])
	*/
	EXPORT int _RowWiseSum(MemoryBuffer& x, const MemoryTile& A, MemoryBuffer& cache, const MatrixOperation aOperation)
	{
		if (cache.size != A.nCols)
		{
			if (cache.pointer != 0)
				_Free(cache);
			cache.pointer = 0;
		}

		if (cache.pointer == 0)
		{
			cache = MemoryBuffer(cache.pointer, A.nCols, A.memorySpace, A.mathDomain);
			_Alloc(cache);
			_Initialize(cache, 1.0);
		}

		return _Dot(x, A, cache, aOperation);
	}
	EXPORT int _RowWiseSumRaw(const ptr_t x, const ptr_t A, const unsigned nRows, const unsigned nCols, const MemorySpace memorySpace, const MathDomain mathDomain, const ptr_t cache, const MatrixOperation aOperation)
	{
		MemoryBuffer _x(x, nRows, memorySpace, mathDomain);
		MemoryTile _A(A, nRows, nCols, memorySpace, mathDomain);
		MemoryBuffer _cache(cache, nCols, memorySpace, mathDomain);
		return _RowWiseSum(_x, _A, _cache, aOperation);
	}

	/**
	* x = sum(A[:, :, ])
	*/
	EXPORT int _CubeWiseSum(MemoryTile& A, const MemoryCube& T, MemoryCube& cacheReshape, MemoryBuffer& cacheOnes)
	{
		if (cacheOnes.size != T.nCubes)
		{
			if (cacheOnes.pointer != 0)
				_Free(cacheOnes);
			cacheOnes.pointer = 0;
		}

		if (cacheOnes.pointer == 0)
		{
			cacheOnes = MemoryBuffer(0, T.nCubes, T.memorySpace, T.mathDomain);
			_Alloc(cacheOnes);
			_Initialize(cacheOnes, 1.0);
		}

		// reshape T into nCols blocks of [nRows * nCubes]
		if (cacheReshape.nRows != T.nRows || cacheReshape.nCols != T.nCubes || cacheReshape.nCubes != T.nCols)
		{
			if (cacheReshape.pointer != 0)
				_Free(cacheReshape);
			cacheReshape.pointer = 0;
		}

		if (cacheReshape.pointer == 0)
		{
			cacheReshape = MemoryCube(0, T.nRows, T.nCubes, T.nCols, T.memorySpace, T.mathDomain);
			_Alloc(cacheReshape);
		}

		const hipblasHandle_t& handle = detail::CublasHandle();
		switch (A.mathDomain)
		{
			case MathDomain::Float:
			{
				dim3 blockDim(32, 32);
				dim3 gridDim((cacheReshape.nRows + 32 - 1) / 32, (cacheReshape.nCols + 32 - 1) / 32);
				CUDA_CALL_XY(__Reshape__<float>, gridDim, blockDim, (float*)cacheReshape.pointer, (float*)T.pointer, T.nRows, T.nCols, T.nCubes);
				break;
			}
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__Reshape__<double>, (double*)cacheReshape.pointer, (double*)T.pointer, T.nRows, T.nCols, T.nCubes);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}

		int err = hipGetLastError();
		if (err)
			return err;

		MemoryCube tmp1(A.pointer, A.nRows, 1, T.nCubes, A.memorySpace, A.mathDomain);
		MemoryCube tmp2(cacheReshape.pointer, cacheReshape.nRows, cacheReshape.nCols, 0, A.memorySpace, A.mathDomain);
		MemoryCube tmp3(cacheOnes.pointer, cacheOnes.size, 0, 0, A.memorySpace, A.mathDomain);
		return _BatchedMultiply(tmp1, tmp2, tmp3,cacheReshape.nRows * cacheReshape.nCols, 0,MatrixOperation::None, MatrixOperation::None, 1.0, 0.0);
	}
	EXPORT int _CubeWiseSumRaw(const ptr_t A, const ptr_t T, const unsigned nRows, const unsigned nCols, const unsigned nCubes, const MemorySpace memorySpace, const MathDomain mathDomain, const ptr_t cacheReshape, const ptr_t cacheOnes)
	{
		MemoryTile _A(A, nRows, nCols, memorySpace, mathDomain);
		MemoryCube _T(T, nRows, nCols, nCubes, memorySpace, mathDomain);
		MemoryCube _cacheReshape(cacheReshape, nRows, nCubes, nCols, memorySpace, mathDomain);
		MemoryBuffer _cacheOnes(cacheOnes, nCubes, memorySpace, mathDomain);
		return _CubeWiseSum(_A, _T, _cacheReshape, _cacheOnes);
	}

	/**
	* X such that A * X = b by means of LU factorization
	*/
	EXPORT int _Solve(const MemoryTile& A, MemoryTile& B, const MatrixOperation aOperation, const LinearSystemSolverType solver)
	{
		const auto& handle = detail::CuSolverHandle();
		const auto& cublasHandle = detail::CublasHandle();

		int err;

		int *info = nullptr;

		switch (A.mathDomain)
		{
			case MathDomain::Float:
			{
				float *buffer = nullptr;

				// Need to copy A, as it will be overwritten by its factorization
				float *aPtr = nullptr;
				err = hipMalloc(&aPtr, A.nRows * A.nRows * sizeof(float));
				if (err)
					return err;
				hipMemcpy(aPtr, (float *) A.pointer, A.nRows * A.nRows * sizeof(float), hipMemcpyDeviceToDevice);

				// calculate buffer size required by the solver
				int bufferSize = 0;
				switch (solver)
				{
					case LinearSystemSolverType::Lu:
						if (hipsolverDnSgetrf_bufferSize(handle, A.nRows, A.nRows, aPtr, A.leadingDimension, &bufferSize))
							return CudaKernelException::_InternalException;
						break;
					case LinearSystemSolverType::Qr:
						if (hipsolverDnSgeqrf_bufferSize(handle, A.nRows, A.nRows, aPtr, A.leadingDimension, &bufferSize))
							return CudaKernelException::_InternalException;
						break;
					default:
						return CudaKernelException::_NotImplementedException;
				}
				err = hipMalloc(&buffer, bufferSize * sizeof(float));
				if (err)
					return err;

				// Initializes auxliary value for solver
				err = hipMalloc(&info, sizeof(int));
				if (err)
					return err;
				err = hipMemset(info, 0, sizeof(int));
				if (err)
					return err;

				// allocate memory for pivoting
				switch (solver)
				{
					case LinearSystemSolverType::Lu:
					{
						int *ipiv = nullptr;

						err = hipMalloc(&ipiv, A.nRows * sizeof(int));
						if (err)
							return err;

						// Factorize A (and overwrite it with L)
						if (hipsolverDnSgetrf(handle, A.nRows, A.nRows, aPtr, A.leadingDimension, buffer, ipiv, info))
							return CudaKernelException::_InternalException;

						// Solve
						err = hipsolverDnSgetrs(handle, cublasOperation[static_cast<unsigned>(aOperation)], A.nRows, B.nCols, aPtr, A.leadingDimension, ipiv, (float *) B.pointer, B.leadingDimension, info);

						hipFree(ipiv);

						break;
					}
					case LinearSystemSolverType::Qr:
					{
						float *tau = nullptr;
						err = hipMalloc((void **) &tau, sizeof(float) * A.nRows);
						if (err)
							return err;

						// compute QR factorization
						if (hipsolverDnSgeqrf(handle, A.nRows, A.nRows, aPtr, A.leadingDimension, tau, buffer, bufferSize, info))
							return CudaKernelException::_InternalException;

						// Q^T * B
						err = hipsolverDnSormqr(
								handle,
								HIPBLAS_SIDE_LEFT,
								HIPBLAS_OP_T,
								A.nRows,
								B.nCols,
								A.nRows,
								aPtr,
								A.leadingDimension,
								tau,
								(float *) B.pointer,
								B.leadingDimension,
								buffer,
								bufferSize,
								info);
						if (err)
							return err;

						// Solve (x = R \ (Q^T * B))
						static constexpr float one = 1.0f;
						err = hipblasStrsm(
								cublasHandle,
								HIPBLAS_SIDE_LEFT,
								HIPBLAS_FILL_MODE_UPPER,
								cublasOperation[static_cast<unsigned>(aOperation)],
								HIPBLAS_DIAG_NON_UNIT,
								A.nRows,
								B.nCols,
								&one,
								aPtr,
								A.leadingDimension,
								(float *)B.pointer,
								B.leadingDimension);
						break;
					}
					default:
						return CudaKernelException::_NotImplementedException;
				}

				hipFree(buffer);
				hipFree(aPtr);
				break;
			}
			case MathDomain::Double:
			{
				double *buffer = nullptr;

				// Need to copy A, as it will be overwritten by its factorization
				double *aPtr = nullptr;
				err = hipMalloc(&aPtr, A.nRows * A.nRows * sizeof(double));
				if (err)
					return err;
				hipMemcpy(aPtr, (double *) A.pointer, A.nRows * A.nRows * sizeof(double), hipMemcpyDeviceToDevice);

				// calculate buffer size required by the solver
				int bufferSize = 0;
				switch (solver)
				{
					case LinearSystemSolverType::Lu:
						if (hipsolverDnDgetrf_bufferSize(handle, A.nRows, A.nRows, aPtr, A.leadingDimension, &bufferSize))
							return CudaKernelException::_InternalException;
						break;
					case LinearSystemSolverType::Qr:
						if (hipsolverDnDgeqrf_bufferSize(handle, A.nRows, A.nRows, aPtr, A.leadingDimension, &bufferSize))
							return CudaKernelException::_InternalException;
						break;
					default:
						return CudaKernelException::_NotImplementedException;
				}

				err = hipMalloc(&buffer, bufferSize * sizeof(double));
				if (err)
					return err;

				// Initializes auxliary value for solver
				err = hipMalloc(&info, sizeof(int));
				if (err)
					return err;
				err = hipMemset(info, 0, sizeof(int));
				if (err)
					return err;

				// allocate memory for pivoting
				switch (solver)
				{
					case LinearSystemSolverType::Lu:
					{
						int *ipiv = nullptr;

						err = hipMalloc(&ipiv, A.nRows * sizeof(int));
						if (err)
							return err;

						// Factorize A (and overwrite it with L)
						if (hipsolverDnDgetrf(handle, A.nRows, A.nRows, aPtr, A.leadingDimension, buffer, ipiv, info))
							return CudaKernelException::_InternalException;

						// Solve
						err = hipsolverDnDgetrs(handle, cublasOperation[static_cast<unsigned>(aOperation)], A.nRows, B.nCols, aPtr, A.leadingDimension, ipiv, (double *) B.pointer, B.leadingDimension, info);
						break;
					}
					case LinearSystemSolverType::Qr:
					{
						double *tau = nullptr;
						err = hipMalloc((void **) &tau, sizeof(double) * A.nRows);
						if (err)
							return err;

						// compute QR factorization
						if (hipsolverDnDgeqrf(handle, A.nRows, A.nRows, aPtr, A.leadingDimension, tau, buffer, bufferSize, info))
							return CudaKernelException::_InternalException;

						// B = Q^T * B
						err = hipsolverDnDormqr(
								handle,
								HIPBLAS_SIDE_LEFT,
								HIPBLAS_OP_T,
								A.nRows,
								B.nCols,
								A.nRows,
								aPtr,
								A.leadingDimension,
								tau,
								(double *) B.pointer,
								B.leadingDimension,
								buffer,
								bufferSize,
								info);
						if (err)
							return err;

						// Solve (x = R \ (Q^T * B))
						static constexpr double one = 1.0;
						err = hipblasDtrsm(
								cublasHandle,
								HIPBLAS_SIDE_LEFT,
								HIPBLAS_FILL_MODE_UPPER,
								cublasOperation[static_cast<unsigned>(aOperation)],
								HIPBLAS_DIAG_NON_UNIT,
								A.nRows,
								B.nCols,
								&one,
								aPtr,
								A.leadingDimension,
								(double *) B.pointer,
								B.leadingDimension);
						break;
					}
					default:
						return CudaKernelException::_NotImplementedException;
				}

				hipFree(buffer);
				hipFree(aPtr);
				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipDeviceSynchronize();

		// free memory
		hipFree(info);

		return err;
	}
	EXPORT int _SolveRaw(const ptr_t A, const ptr_t B, const unsigned nRows, const unsigned nCols, const MemorySpace memorySpace, const MathDomain mathDomain, const MatrixOperation aOperation)
	{
		MemoryTile _A(A, nRows, nCols, memorySpace, mathDomain);
		MemoryTile _B(B, nRows, nCols, memorySpace, mathDomain);
		return _Solve(_A, _B, aOperation);
	}

	/**
	* A = A^(-1) by means of LU factorization
	*/
	EXPORT int _Invert(MemoryTile& A, const MatrixOperation aOperation)
	{
		float* eyePtr = nullptr;
		int err = hipMalloc(&eyePtr, A.TotalSize());
		if (err)
			return err;
		MemoryTile eye((ptr_t)eyePtr, A.nRows, A.nRows, A.memorySpace, A.mathDomain);
		err = _Eye(eye);
		if (err)
			return err;
		err = _Solve(A, eye, aOperation);
		if (err)
			return err;

		// This might not be the fastest implementation, but it's general enough
		switch (A.mathDomain)
		{
		case MathDomain::Float:
		{
			err = hipMemcpy((float*)A.pointer, (float*)eye.pointer, A.TotalSize(), hipMemcpyDefault);
			if (err)
				return err;
			break;
		}
		case MathDomain::Double:
		{
			err = hipMemcpy((double*)A.pointer, (double*)eye.pointer, A.TotalSize(), hipMemcpyDefault);
			if (err)
				return err;
			break;
		}
		default:
			return CudaKernelException::_NotImplementedException;
		}

		hipFree((void*)eye.pointer);
		return err;
	}
	EXPORT int _InvertRaw(const ptr_t A, const unsigned nRows, const unsigned nCols, const MemorySpace memorySpace, const MathDomain mathDomain, const MatrixOperation aOperation)
	{
		MemoryTile _A(A, nRows, nCols, memorySpace, mathDomain);
		return _Invert(_A, aOperation);
	}

	EXPORT int _ArgAbsMin(int& argMin, const MemoryBuffer& x)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		int err = 0;
		switch (x.mathDomain)
		{
			case MathDomain::Float:
				err = hipblasIsamin(handle, x.size, (float*)x.pointer, 1, &argMin);
				break;
			case MathDomain::Double:
				err = hipblasIdamin(handle, x.size, (double*)x.pointer, 1, &argMin);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}

		if (err)
			return err;

		// cublasI<t>amin uses 1-indexed array
		--argMin;
		return hipGetLastError();
	}

	EXPORT int _ColumnWiseArgAbsMin(MemoryBuffer& argMin, const MemoryTile& A)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

		int err = 0;
		switch (A.mathDomain)
		{
			case MathDomain::Float:
				for (size_t j = 0; j < A.nCols; ++j)
					err = hipblasIsamin(handle, A.nRows, (float*)(A.pointer + j * A.nRows * A.ElementarySize()), 1, (int*)(argMin.pointer + j * argMin.ElementarySize()));
				break;
			case MathDomain::Double:
				for (size_t j = 0; j < A.nCols; ++j)
					err = hipblasIdamin(handle, A.nRows, (double*)(A.pointer + j * A.nRows * A.ElementarySize()), 1, (int*)(argMin.pointer + j * argMin.ElementarySize()));
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

		if (err)
			return err;

		return hipGetLastError();
	}

	EXPORT int _ArgAbsMax(int& argMax, const MemoryBuffer& x)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		int err = 0;
		switch (x.mathDomain)
		{
			case MathDomain::Float:
				err = hipblasIsamax(handle, x.size, (float*)x.pointer, 1, &argMax);
				break;
			case MathDomain::Double:
				err = hipblasIdamax(handle, x.size, (double*)x.pointer, 1, &argMax);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}

		if (err)
			return err;

		// cublasI<t>amax uses 1-indexed array
		--argMax;
		return hipGetLastError();
	}

	EXPORT int _ColumnWiseArgAbsMax(MemoryBuffer& argMax, const MemoryTile& A)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

		int err = 0;
		switch (A.mathDomain)
		{
			case MathDomain::Float:
				for (size_t j = 0; j < A.nCols; ++j)
					err = hipblasIsamax(handle, A.nRows, (float*)(A.pointer + j * A.nRows * A.ElementarySize()), 1, (int*)(argMax.pointer + j * argMax.ElementarySize()));
				break;
			case MathDomain::Double:
				for (size_t j = 0; j < A.nCols; ++j)
					err = hipblasIdamax(handle, A.nRows, (double*)(A.pointer + j * A.nRows * A.ElementarySize()), 1, (int*)(argMax.pointer + j * argMax.ElementarySize()));
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

		if (err)
			return err;

		return hipGetLastError();
	}

	EXPORT int _IsNonZero(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__IsNonZero__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__IsNonZero__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			case MathDomain::Int:
				CUDA_CALL_SINGLE(__IsNonZero__<int>, (int*)z.pointer, (int*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}

		return hipGetLastError();
	}

	EXPORT int _EuclideanNorm(double& norm, const MemoryBuffer& x)
	{
		const hipblasHandle_t& handle = detail::CublasHandle();

		switch (x.mathDomain)
		{
			case MathDomain::Float:
			{
				auto _norm = (float)norm;
				int err = hipblasSnrm2(handle, x.size, (float*)x.pointer, 1, &_norm);

				norm = _norm;
				return err;
			}
			case MathDomain::Double:
				return hipblasDnrm2(handle, x.size, (double*)x.pointer, 1, &norm);
			case MathDomain::Int:
			default:
				return CudaKernelException::_NotImplementedException;
		}
	}
}

GLOBAL void __IntAffineOperationNaive__(int* z, const int* x, const int* y, const size_t sz, const int alpha, const int beta, const int gamma)
{
	CUDA_FUNCTION_PROLOGUE
	CUDA_FOR_LOOP_PROLOGUE

		z[i] = alpha * x[i] + beta * y[i] + gamma * z[i];

	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T, bool inverse>
GLOBAL void __ElementwiseProductNaive__(T* RESTRICT z, const T* RESTRICT x, const T* RESTRICT y, const size_t sz, const T alpha)
{
	CUDA_FUNCTION_PROLOGUE
	CUDA_FOR_LOOP_PROLOGUE

		z[i] = x[i] * (!inverse ? y[i] : (static_cast<T>(1.0) / y[i])) * alpha;

	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __IsNonZero__(T* RESTRICT z, const T* RESTRICT x, const size_t sz)
{
	CUDA_FUNCTION_PROLOGUE
	CUDA_FOR_LOOP_PROLOGUE

		z[i] = (x[i] > static_cast<T>(1e-12) || x[i] < static_cast<T>(-1e-12)) ? 1 : 0;

	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __Reshape__(T* RESTRICT out, const T* RESTRICT in, const size_t nRows, const size_t nCols, const size_t nCubes)
{
	const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int stepX = gridDim.x * blockDim.x;

	const int tidY = blockDim.y * blockIdx.y + threadIdx.y;
	const unsigned int stepY = gridDim.y * blockDim.y;

	const size_t inMatrixSize = nRows * nCols;
	const size_t outMatrixSize = nRows * nCubes;
	for (size_t i = tidX; i < nRows; i += stepX)
	{
		for (size_t j = tidY; j < nCols; j += stepY)
		{
			const size_t inStride = i + j * nRows;
			const size_t outOffset = i + j * outMatrixSize;
			for (size_t k = 0; k < nCubes; ++k)
				out[outOffset + k * nRows] = in[inStride + k * inMatrixSize];
		}
	}
}
