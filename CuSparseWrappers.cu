#include "CuSparseWrappers.cuh"
#include "DeviceManager.cuh"

EXTERN_C
{
	/**
	* zDense = alpha * xSparse + yDense
	*/
	EXPORT int _SparseAdd(MemoryBuffer z, const SparseMemoryBuffer x, const MemoryBuffer y, const double alpha)
	{
		const hipsparseHandle_t& cuSparseHandle = detail::CuSparseHandle();
		const hipblasHandle_t& cuBlasHandle = detail::CublasHandle();

		int err = -1;
		switch (z.mathDomain)
		{
		case MathDomain::Float:
		{
			if (hipblasScopy(cuBlasHandle, z.size, (float*)y.pointer, 1, (float*)z.pointer, 1))
				return CudaKernelException::_InternalException;

			const float _alpha = (float)alpha;
			err = hipsparseSaxpyi(cuSparseHandle, x.size, &_alpha, (float*)x.pointer, (int*)x.indices, (float*)z.pointer, HIPSPARSE_INDEX_BASE_ZERO);
			break;
		};
		case MathDomain::Double:
		{
			if (hipblasDcopy(cuBlasHandle, z.size, (double*)y.pointer, 1, (double*)z.pointer, 1))
				return CudaKernelException::_InternalException;

			err = hipsparseDaxpyi(cuSparseHandle, x.size, &alpha, (double*)x.pointer, (int*)x.indices, (double*)z.pointer, HIPSPARSE_INDEX_BASE_ZERO);
			break;
		};;
		default: 
			return CudaKernelException::_NotImplementedException;
		}
		
		hipDeviceSynchronize(); // axpy is asynch!

		if (err)
			return err;
		return hipGetLastError();
	}

	/**
	*	yDense = ASparse * xDense
	*/
	EXPORT int _SparseDot(MemoryBuffer y, const SparseMemoryTile A, const MemoryBuffer x, const MatrixOperation aOperation, const double alpha)
	{
		const hipsparseHandle_t& handle = detail::CuSparseHandle();
		const hipsparseMatDescr_t& descr = detail::CsrMatrixDescription();

		int err = -1;

		switch (y.mathDomain)
		{
		case MathDomain::Float:
		{
			const float beta = 0.0f;
			const float _alpha = (float)alpha;

			err = hipsparseScsrmv(handle, cusparseOperation[static_cast<int>(aOperation)],
				A.nRows, A.nCols, A.size,
				&_alpha, descr,
				(float*)A.pointer, (int*)A.nNonZeroRows, (int*)A.nonZeroColumnIndices,
				(float*)x.pointer,
				&beta,
				(float*)y.pointer);
			break;
		};
		case MathDomain::Double:
		{
			const double beta = 0.0;

			err = hipsparseDcsrmv(handle, cusparseOperation[static_cast<int>(aOperation)],
				A.nRows, A.nCols, A.size,
				&alpha, descr,
				(double*)A.pointer, (int*)A.nNonZeroRows, (int*)A.nonZeroColumnIndices,
				(double*)x.pointer,
				&beta,
				(double*)y.pointer);
			break;
		};;
		default:
			return CudaKernelException::_NotImplementedException;
		}

		if (err)
			return err;
		return hipGetLastError();
	}

	/**
	*	ADense = BSparse * CDense
	*/
	EXPORT int _SparseMultiply(MemoryTile A, const SparseMemoryTile B, const MemoryTile C, const unsigned leadingDimensionB, const unsigned leadingDimensionC, const MatrixOperation bOperation, const double alpha)
	{
		const hipsparseHandle_t& handle = detail::CuSparseHandle();
		const hipsparseMatDescr_t& descr = detail::CsrMatrixDescription();

		int err = -1;

		switch (A.mathDomain)
		{
		case MathDomain::Float:
		{
			const float beta = 0.0f;
			const float _alpha = (float)alpha;

			err = hipsparseScsrmm(handle, cusparseOperation[static_cast<int>(bOperation)],
				leadingDimensionB, C.nCols, leadingDimensionC, B.nNonZeroRows,
				&_alpha,
				descr, (float*)B.pointer, (int*)B.nNonZeroRows, (int*)B.nonZeroColumnIndices,
				(float*)C.pointer, leadingDimensionC,
				&beta,
				(float*)A.pointer, leadingDimensionB);
			break;
		}
		case MathDomain::Double:
		{
			const double beta = 0.0;

			err = hipsparseDcsrmm(handle, cusparseOperation[static_cast<int>(bOperation)],
				leadingDimensionB, C.nCols, leadingDimensionC, B.nNonZeroRows,
				&alpha,
				descr, (double*)B.pointer, (int*)B.nNonZeroRows, (int*)B.nonZeroColumnIndices,
				(double*)C.pointer, leadingDimensionC,
				&beta,
				(double*)A.pointer, leadingDimensionB);
			break;
		}
		default:
			return CudaKernelException::_NotImplementedException;
		}

		if (err)
			return err;
		return hipGetLastError();
	}

}