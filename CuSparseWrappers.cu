#include "CuSparseWrappers.cuh"
#include "DeviceManager.cuh"

EXTERN_C
{
	/**
	* zDense = alpha * xSparse + yDense
	*/
	EXPORT int _SparseAdd(MemoryBuffer z, const SparseMemoryBuffer x, const MemoryBuffer y, const double alpha)
	{
		const hipsparseHandle_t& cuSparseHandle = detail::CuSparseHandle();
		const hipblasHandle_t& cuBlasHandle = detail::CublasHandle();

		switch (z.mathDomain)
		{
		case MathDomain::Float:
		{
			const int err = hipblasScopy(cuBlasHandle, z.size, (float*)y.pointer, 1, (float*)z.pointer, 1);
			if (err)
				return err;

			const float _alpha = (float)alpha;
			hipsparseSaxpyi(cuSparseHandle, x.nNonZeros, &_alpha, (float*)x.pointer, (int*)x.indices, (float*)z.pointer, HIPSPARSE_INDEX_BASE_ZERO);
			break;
		};
		case MathDomain::Double:
		{
			const int err = hipblasDcopy(cuBlasHandle, z.size, (double*)y.pointer, 1, (double*)z.pointer, 1);
			if (err)
				return err;

			hipsparseDaxpyi(cuSparseHandle, x.nNonZeros, &alpha, (double*)x.pointer, (int*)x.indices, (double*)z.pointer, HIPSPARSE_INDEX_BASE_ZERO);
			break;
		};;
		default: 
			return -1;
		}
		
		hipDeviceSynchronize(); // axpy is asynch!
		return hipGetLastError();
	}

	/**
	*	yDense = ASparse * xDense
	*/
	EXPORT int _SparseDot(MemoryBuffer y, const SparseMemoryTile A, const MemoryBuffer x, const double alpha)
	{
		const hipsparseHandle_t& handle = detail::CuSparseHandle();
		const hipsparseMatDescr_t& descr = detail::CsrMatrixDescription();

		int err = -1;

		switch (y.mathDomain)
		{
		case MathDomain::Float:
		{
			const float beta = 0.0f;
			const float _alpha = (float)alpha;

			err = hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				A.nRows, A.nCols, A.nNonZeros,
				&_alpha, descr,
				(float*)A.pointer, (int*)A.nNonZeroRows, (int*)A.nonZeroColumnIndices,
				(float*)x.pointer,
				&beta,
				(float*)y.pointer);
			break;
		};
		case MathDomain::Double:
		{
			const double beta = 0.0;

			err = hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				A.nRows, A.nCols, A.nNonZeros,
				&alpha, descr,
				(double*)A.pointer, (int*)A.nNonZeroRows, (int*)A.nonZeroColumnIndices,
				(double*)x.pointer,
				&beta,
				(double*)y.pointer);
			break;
		};;
		default:
			return -1;
		}

		if (err)
			return err;
		return hipGetLastError();
	}

	/**
	*	ADense = BSparse * CDense
	*/
	EXPORT int _SparseMultiply(MemoryTile A, const SparseMemoryTile B, const MemoryTile C, const unsigned leadingDimensionB, const unsigned leadingDimensionC, const double alpha)
	{
		const hipsparseHandle_t& handle = detail::CuSparseHandle();
		const hipsparseMatDescr_t& descr = detail::CsrMatrixDescription();

		int err = -1;

		switch (A.mathDomain)
		{
		case MathDomain::Float:
		{
			const float beta = 0.0f;
			const float _alpha = (float)alpha;

			err = hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				leadingDimensionB, C.nCols, leadingDimensionC, B.nNonZeroRows,
				&_alpha,
				descr, (float*)B.pointer, (int*)B.nNonZeroRows, (int*)B.nonZeroColumnIndices,
				(float*)C.pointer, leadingDimensionC,
				&beta,
				(float*)A.pointer, leadingDimensionB);
			break;
		}
		case MathDomain::Double:
		{
			const double beta = 0.0;

			err = hipsparseDcsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				leadingDimensionB, C.nCols, leadingDimensionC, B.nNonZeroRows,
				&alpha,
				descr, (double*)B.pointer, (int*)B.nNonZeroRows, (int*)B.nonZeroColumnIndices,
				(double*)C.pointer, leadingDimensionC,
				&beta,
				(double*)A.pointer, leadingDimensionB);
			break;
		}
		default:
			return -1;
		}

		if (err)
			return err;
		return hipGetLastError();
	}

}