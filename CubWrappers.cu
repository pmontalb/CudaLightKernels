#include "hip/hip_runtime.h"
#include "CubWrappers.cuh"
#include "MemoryManager.cuh"

struct Abs
{
	template <typename T>
	HOST DEVICE __forceinline__ T operator()(const T &a) const
	{
		return a > 0 ? a : -a;
	}
};

struct AbsMin
{
	template <typename T>
	HOST DEVICE __forceinline__ T operator()(const T &a, const T &b) const
	{
		return CUB_MIN(Abs()(a), Abs()(b));
	}
};

struct AbsMax
{
	template <typename T>
	HOST DEVICE __forceinline__ T operator()(const T &a, const T &b) const
	{
		return CUB_MAX(Abs()(a), Abs()(b));
	}
};

EXTERN_C
{
	EXPORT int _Sum(double& sum, const MemoryBuffer v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				float _sum;
				hipMemcpy(&_sum, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				sum = _sum;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				hipMemcpy(&sum, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				int _sum;
				hipMemcpy(&_sum, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				sum = _sum;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}

	EXPORT int _Min(double& min, const MemoryBuffer v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				float _min;
				hipMemcpy(&_min, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				min = _min;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				hipMemcpy(&min, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				int _min;
				hipMemcpy(&_min, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				min = _min;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}

	EXPORT int _Max(double& max, const MemoryBuffer v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				float _max;
				hipMemcpy(&_max, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				max = _max;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				hipMemcpy(&max, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				int _max;
				hipMemcpy(&_max, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				max = _max;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}

	EXPORT int _AbsMin(double& min, const MemoryBuffer v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size, ::AbsMin(), 1e9);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size, ::AbsMin(), 1e9);

				float _min;
				hipMemcpy(&_min, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				min = _min;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size, ::AbsMin(), 1e9);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size, ::AbsMin(), 1e9);

				hipMemcpy(&min, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size, ::AbsMin(), 1e9);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size, ::AbsMin(), 1e9);

				int _min;
				hipMemcpy(&_min, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				min = _min;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}

	EXPORT int _AbsMax(double& max, const MemoryBuffer v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size, ::AbsMax(), 0);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size, ::AbsMax(), 0);

				float _max;
				hipMemcpy(&_max, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				max = _max;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size, ::AbsMax(), 0);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size, ::AbsMax(), 0);

				hipMemcpy(&max, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size, ::AbsMax(), 0);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size, ::AbsMax(), 0);

				int _max;
				hipMemcpy(&_max, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				max = _max;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}
}