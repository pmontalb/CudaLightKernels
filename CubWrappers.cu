#include "CubWrappers.cuh"
#include "MemoryManager.cuh"

EXTERN_C
{
	EXPORT int _Sum(double& sum, const MemoryBuffer v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				float _sum;
				hipMemcpy(&_sum, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				sum = _sum;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				hipMemcpy(&sum, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Sum(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				int _sum;
				hipMemcpy(&_sum, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				sum = _sum;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}
}