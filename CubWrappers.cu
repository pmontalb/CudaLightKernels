#include "hip/hip_runtime.h"
#include "CubWrappers.cuh"
#include "MemoryManager.cuh"

struct Abs
{
	template <typename T>
	HOST DEVICE __forceinline__ T operator()(const T &a) const
	{
		return a > 0 ? a : -a;
	}
};

struct AbsMin
{
	template <typename T>
	HOST DEVICE __forceinline__ T operator()(const T &a, const T &b) const
	{
		return CUB_MIN(Abs()(a), Abs()(b));
	}
};

struct AbsMax
{
	template <typename T>
	HOST DEVICE __forceinline__ T operator()(const T &a, const T &b) const
	{
		return CUB_MAX(Abs()(a), Abs()(b));
	}
};

EXTERN_C
{
	EXPORT int _Sum(double& sum, const MemoryBuffer& v)
	{
		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		MemoryBuffer cacheBuffer;
		_DetermineSumCache(cacheBuffer, v, output);
		
		auto ret = _SumWithProvidedCache(sum, v, cacheBuffer, output);
		_Free(cacheBuffer);
		_Free(output);

		return hipGetLastError();
	}

	EXPORT int _DetermineSumCache(MemoryBuffer& cacheBuffer, const MemoryBuffer& v, const MemoryBuffer& oneElementCache)
	{
		void* cache = nullptr;
		size_t cacheSize = 0;
		
		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Sum(cache, cacheSize, (float*)v.pointer, (float*)oneElementCache.pointer, v.size);
				
				// Allocate temporary storage
				hipMalloc(&cache, cacheSize);
				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Sum(cache, cacheSize, (double*)v.pointer, (double*)oneElementCache.pointer, v.size);
				
				// Allocate temporary storage
				hipMalloc(&cache, cacheSize);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Sum(cache, cacheSize, (int*)v.pointer, (int*)oneElementCache.pointer, v.size);
				
				// Allocate temporary storage
				hipMalloc(&cache, cacheSize);
				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}
		
		cacheBuffer.pointer = reinterpret_cast<ptr_t>(cache);
		cacheBuffer.size = static_cast<unsigned>(cacheSize);
		cacheBuffer.memorySpace = MemorySpace::Device;
		cacheBuffer.mathDomain = v.mathDomain;
		
		return hipGetLastError();
	}

	EXPORT int _SumWithProvidedCache(double& sum, const MemoryBuffer& v, MemoryBuffer& cache, MemoryBuffer& outputCache)
	{
		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				// Run sum-reduction
				size_t totalSize = cache.size;
				hipcub::DeviceReduce::Sum((void*)cache.pointer, totalSize, (float*)v.pointer, (float*)outputCache.pointer, v.size);
				
				float _sum;
				hipMemcpy(&_sum, (float*)outputCache.pointer, sizeof(float), hipMemcpyDeviceToHost);
				sum = _sum;
				
				break;
			}
			case MathDomain::Double:
			{
				// Run sum-reduction
				size_t totalSize = cache.size;
				hipcub::DeviceReduce::Sum((void*)cache.pointer, totalSize, (double*)v.pointer, (double*)outputCache.pointer, v.size);
				
				hipMemcpy(&sum, (double*)outputCache.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				// Run sum-reduction
				size_t totalSize = cache.size;
				hipcub::DeviceReduce::Sum((void*)cache.pointer, totalSize, (int*)v.pointer, (int*)outputCache.pointer, v.size);
				
				int _sum;
				hipMemcpy(&_sum, (int*)outputCache.pointer, sizeof(int), hipMemcpyDeviceToHost);
				sum = _sum;
				
				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}
		
		return hipGetLastError();
	}

	EXPORT int _Min(double& min, const MemoryBuffer& v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				float _min;
				hipMemcpy(&_min, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				min = _min;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				hipMemcpy(&min, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				int _min;
				hipMemcpy(&_min, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				min = _min;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}

	EXPORT int _Max(double& max, const MemoryBuffer& v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size);

				float _max;
				hipMemcpy(&_max, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				max = _max;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Min(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size);

				hipMemcpy(&max, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Max(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size);

				int _max;
				hipMemcpy(&_max, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				max = _max;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}

	EXPORT int _AbsMin(double& min, const MemoryBuffer& v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size, ::AbsMin(), 1e9);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size, ::AbsMin(), 1e9);

				float _min;
				hipMemcpy(&_min, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				min = _min;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size, ::AbsMin(), 1e9);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size, ::AbsMin(), 1e9);

				hipMemcpy(&min, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size, ::AbsMin(), 1e9);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size, ::AbsMin(), 1e9);

				int _min;
				hipMemcpy(&_min, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				min = _min;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}

	EXPORT int _AbsMax(double& max, const MemoryBuffer& v)
	{
		void* cache = nullptr;

		MemoryBuffer output(0, 1, v.memorySpace, v.mathDomain);
		// Determine temporary device storage requirements
		_Alloc(output);

		size_t temp_storage_bytes = 0;

		switch (v.mathDomain)
		{
			case MathDomain::Float:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size, ::AbsMax(), 0);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (float*)v.pointer, (float*)output.pointer, v.size, ::AbsMax(), 0);

				float _max;
				hipMemcpy(&_max, (float*)output.pointer, sizeof(float), hipMemcpyDeviceToHost);
				max = _max;

				break;
			}
			case MathDomain::Double:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size, ::AbsMax(), 0);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (double*)v.pointer, (double*)output.pointer, v.size, ::AbsMax(), 0);

				hipMemcpy(&max, (double*)output.pointer, sizeof(double), hipMemcpyDeviceToHost);
				break;
			}
			case MathDomain::Int:
			{
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size, ::AbsMax(), 0);

				// Allocate temporary storage
				hipMalloc(&cache, temp_storage_bytes);

				// Run sum-reduction
				hipcub::DeviceReduce::Reduce(cache, temp_storage_bytes, (int*)v.pointer, (int*)output.pointer, v.size, ::AbsMax(), 0);

				int _max;
				hipMemcpy(&_max, (int*)output.pointer, sizeof(int), hipMemcpyDeviceToHost);
				max = _max;

				break;
			}
			default:
				return CudaKernelException::_NotImplementedException;
		}

		hipFree(cache);
		_Free(output);

		return hipGetLastError();
	}
}