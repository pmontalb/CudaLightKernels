#include "MemoryManager.cuh"
#include <stdio.h>

EXTERN_C
{
	namespace clk
	{
		EXPORT int _HostToHostCopy(MemoryBuffer dest, const MemoryBuffer source)
		{
			return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyHostToHost);
		}

		EXPORT int _HostToDeviceCopy(MemoryBuffer dest, const MemoryBuffer source)
		{
			return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyHostToDevice);
		}

		EXPORT int _DeviceToHostCopy(MemoryBuffer dest, const MemoryBuffer source)
		{
			return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyDeviceToHost);
		}

		EXPORT int _DeviceToDeviceCopy(MemoryBuffer dest, const MemoryBuffer source)
		{
			return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyDeviceToDevice);
		}

		EXPORT int _AutoCopy(MemoryBuffer dest, const MemoryBuffer source)
		{
			return hipMemcpy((void *)dest.pointer, (void *)source.pointer, dest.TotalSize(), hipMemcpyDefault);
		}

		EXPORT int _Alloc(MemoryBuffer& buf)
		{
			return hipMalloc((void **)&buf.pointer, buf.TotalSize());
		}

		EXPORT int _AllocHost(MemoryBuffer& buf)
		{
			return hipHostMalloc((void **)&buf.pointer, buf.TotalSize());
		}

		EXPORT int _Free(const MemoryBuffer buf)
		{
			hipDeviceSynchronize();
			return hipFree((void *)buf.pointer);
		}

		EXPORT int _FreeHost(const MemoryBuffer buf)
		{
			return hipHostFree((void *)buf.pointer);
		}
	}
}